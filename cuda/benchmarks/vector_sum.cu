#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "vector_sum.cuh"

//////////////////////////////
//////////////////////////////

__global__ void gpu_vector_sum_1(float *x, float *res_tmp, int N) {
    extern __shared__ float shared_data[];
    // each thread loads one element from global to shared mem (warning: no boundary checks!)
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    shared_data[tid] = x[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) res_tmp[blockIdx.x] = shared_data[0];
}


//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;
void VectorSum::alloc() {
    // Compute the number of blocks for implementations where the value is a function of the input size;
    B = (N + block_size_1d - 1) / block_size_1d;

    // Allocate CPU data;
    x = (float*) malloc(sizeof(float) * N);
    res_tmp = (float*) malloc(sizeof(float) * B);
    // Allocate GPU data;
    err = hipMalloc(&x_d, sizeof(float) * N);
    // The GPU output buffer has size equal to the number of blocks, 
    // as we aggregate partial sums on the CPU;
    err = hipMalloc(&res_tmp_d, sizeof(float) * B);
}

// Initialize data;
void VectorSum::init() {
    // Just put some values into the array (sum_{i=1}^{N}{1/i**2} is pi^2 / 6);
    for (int i = 0; i < N; i++) {
        x[i] = float(1) / ((i + 1) * (i + 1));
    }
}

// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void VectorSum::reset() {
    // Reset the result;
    res = 0.0;
    // Transfer data to the GPU;
    hipMemcpy(x_d, x, sizeof(float) * N, hipMemcpyHostToDevice);
}

void VectorSum::vector_sum_1() {
    // Call the GPU computation (and set the size of shared memory!);
    gpu_vector_sum_1<<<B, block_size_1d, sizeof(float) * B>>>(x_d, res_tmp_d, N);
    // Copy the partial result from the GPU to the CPU;
    hipMemcpy(res_tmp, res_tmp_d, sizeof(float) * B, hipMemcpyDeviceToHost);
    // Sum the partial results using the CPU;
    for (int i = 0; i < B; i++) {
        res += res_tmp[i];
    }
}

void VectorSum::execute(int iter) {
    switch (implementation)
    {
    case 0:
        vector_sum_1();
        break;
    default:
        break;
    }
}

#define PI 3.14159265358979323846
void VectorSum::cpu_validation(int iter) {
    float cpu_result = PI * PI / 6;
    if (std::abs(res - cpu_result / 6) > 1e-4) std::cout << "result error! GPU=" << res << ", CPU=" << cpu_result << std::endl; 
}

std::string VectorSum::print_result(bool short_form) {
    return std::to_string(res);
}

void VectorSum::clean() {
    free(x);
    free(res_tmp);
    free(x_d);
    free(res_tmp_d);
}
